#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fcntl.h>
#include <stdint.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

#include "../gpu_usermap_abi.h"

#define DEV_PATH "/dev/gpu_usermap"

#define MAP_LEN 16384

#define unlikely(x)  __builtin_expect(!!(x), 0)

#define ASSERT_CUDA(val)												\
    if(unlikely((val))) {fprintf(stderr, __FILE__, __LINE__, "ERROR: errno = %3d : %s\n", static_cast<int>(val), hipGetErrorString(val)); exit(1);}

void fill_mem(char* addr) {
	for (int i = 0; i < MAP_LEN; i++) {
		addr[i] = 'a' + (i % 26);
	}
}

bool check_mem(char *addr) {
	for (int i = 0; i < MAP_LEN; i++) {
		if (addr[i] != 'a' + (i % 26)) {
			fprintf(stderr, "ERROR: %d th character differs (%c instead of %c)\n",
					i, addr[i], 'a' + (i % 26));
			return false;
		}
	}
	return true;
}

template<int N>
__global__ void gpu_perf(volatile int* mem) {
	unsigned long t1, t2;

	mem[0] = 0;
	t1 = clock64();
	for (int i = 0; i < N; i++) {
		while (mem[0] != 2*i);
		mem[0] = 2*i + 1;
		//__threadfence_system();
	}
	t2 = clock64();
	printf("clock per update: %ld\n", (t2-t1)/N);
}

int hostptr_gpudirect(volatile int** phostptr, volatile int** pdevptr) {
	int ret;
	gpu_usermap_req req;
	volatile int *devptr;

	int fd = open(DEV_PATH, O_RDWR);
	if (fd < 0) {
		perror("open");
		exit(1);
	}

	ASSERT_CUDA(hipMalloc(&devptr, MAP_LEN));
	
	req.magic = GUSERMAP_MAGIC;
	req.gpu_addr = (uintptr_t)devptr;
	req.len = MAP_LEN;

	ret = write(fd, &req, sizeof(req));
	if (ret != sizeof(req)) {
		perror("write");
		fprintf(stderr, "ERROR: write ret: %d expected %d\n", ret, sizeof(req));
		exit(1);
	}
	
	volatile int *addr = (volatile int*)mmap(NULL, MAP_LEN, PROT_WRITE|PROT_READ, MAP_SHARED, fd, 0);
	if (addr == MAP_FAILED) {
		perror("mmap failed");
		exit(1);
	}

	*phostptr = addr;
	*pdevptr = devptr;

	return 0;
}

int hostptr_zerocopy(volatile int** phostptr, volatile int** pdevptr) {

    ASSERT_CUDA(hipHostAlloc((void**)phostptr, MAP_LEN, hipHostMallocMapped));
    ASSERT_CUDA(hipHostGetDevicePointer((void**)pdevptr, (void*)*phostptr, 0));
	
	return 0;
}

void usage() {
	fprintf(stderr, "Usage: ./test_gpu_usermap_perf [case]\n");
	fprintf(stderr, "  case 0 -- gpudirect\n");
	fprintf(stderr, "  case 1 -- zerocopy\n");
}

int main(int argc, char** argv) {
	volatile int* hostptr;
	volatile int *devptr;
	int ret;

	if (argc < 2) {
		usage();
		exit(1);
	}

	if (argv[1][0] == '0') {
		ret = hostptr_gpudirect(&hostptr, &devptr);
	} else if (argv[1][0] == '1') {
		ret = hostptr_zerocopy(&hostptr, &devptr);
	} else {
		usage();
		exit(1);
	}

	if (ret) {
		fprintf(stderr, "hostptr allocation failed ret: %d\n", ret);
		exit(1);
	}

#define NR_ITEMS 1000000

	gpu_perf<NR_ITEMS><<<1, 1>>>(devptr);

	timeval tv1, tv2, tv3;
	for (int i = 0; i < NR_ITEMS; i++) {
		if (i == 1)
			gettimeofday(&tv1, NULL);
			
		while (hostptr[0] != 2*i+1);
		hostptr[0] = 2*i+2;
	}
	gettimeofday(&tv2, NULL);
	
	ASSERT_CUDA(hipDeviceSynchronize());

	munmap((void*)hostptr, MAP_LEN);

	// int idx_second_non_zero = 0, idx_last;

	// for (int i = 0; i < NR_ITEMS; i++) {
	// 	if (arr[i] != 0 && idx_second_non_zero == 0)
	// 		idx_second_non_zero = i+1;
	// 	if (arr[i] >= (NR_ITEMS - 1)) {
	// 		idx_last = i;
	// 		break;
	// 	}
	// }

	// printf("idx difference: %d avg interval: %d\n",
	// 	   (idx_last - idx_second_non_zero),
	// 	   NR_ITEMS / (idx_last - idx_second_non_zero));

	timersub(&tv2, &tv1, &tv3);

	double t_ms = (tv3.tv_sec*1000.0 + tv3.tv_usec/1000.0);
	printf("total time: %.2f ms, for each pingpong: %2f us \n", t_ms, t_ms*1000/(NR_ITEMS-1));
	
	return 0;
}
