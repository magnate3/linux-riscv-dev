#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include "../kernel-module/picoevb-rdma-ioctl.h"

#define SURFACE_W	1024
#define SURFACE_H	1024
#define SURFACE_SIZE	(SURFACE_W * SURFACE_H)

#define OFFSET(x, y)	(((y) * SURFACE_W) + x)
#define DATA(x, y)	(((y & 0xffff) << 16) | ((x) & 0xffff))

extern "C" __global__ void fill_surface(uint32_t *output, uint32_t xor_val)
{
	unsigned int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;

	output[OFFSET(pos_x, pos_y)] = DATA(pos_x, pos_y) ^ xor_val;
}

int main(int argc, char **argv)
{
	hipError_t ce;
	hipError_t cr;
	uint32_t *src_d, *dst_d, *dst_cpu;
	uint32_t y, x;
	int fd, ret;
	unsigned int flag = 1;
	struct picoevb_rdma_pin_cuda pin_params_src, pin_params_dst;
	struct picoevb_rdma_h2c2h_dma dma_params;
	struct picoevb_rdma_unpin_cuda unpin_params_src, unpin_params_dst;

	if (argc != 1) {
		fprintf(stderr, "usage: rdma-cuda\n");
		return 1;
	}

	fd = open("/dev/picoevb", O_RDWR);
	if (fd < 0) {
		perror("open() failed");
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipMalloc(&src_d, SURFACE_SIZE * sizeof(*src_d));
#else
	ce = hipHostAlloc(&src_d, SURFACE_SIZE * sizeof(*src_d),
		hipHostMallocDefault);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Allocation of src_d failed: %d\n", ce);
		return 1;
	}

	cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
		(hipDeviceptr_t)src_d);
	if (cr != hipSuccess) {
		fprintf(stderr, "hipPointerSetAttribute(src_d) failed: %d\n", cr);
		return 1;
	}

	pin_params_src.va = (__u64)src_d;
	pin_params_src.size = SURFACE_SIZE * sizeof(*src_d);
	ret = ioctl(fd, PICOEVB_IOC_PIN_CUDA, &pin_params_src);
	if (ret != 0) {
		fprintf(stderr, "ioctl(PIN_CUDA src) failed: ret=%d errno=%d\n", ret, errno);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipMalloc(&dst_d, SURFACE_SIZE * sizeof(*dst_d));
#else
	ce = hipHostAlloc(&dst_d, SURFACE_SIZE * sizeof(*dst_d),
		hipHostMallocDefault);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Allocation of dst_d failed: %d\n", ce);
		return 1;
	}

	cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
		(hipDeviceptr_t)dst_d);
	if (cr != hipSuccess) {
		fprintf(stderr, "hipPointerSetAttribute(dst_d) failed: %d\n", cr);
		return 1;
	}

	pin_params_dst.va = (__u64)dst_d;
	pin_params_dst.size = SURFACE_SIZE * sizeof(*dst_d);
	ret = ioctl(fd, PICOEVB_IOC_PIN_CUDA, &pin_params_dst);
	if (ret != 0) {
		fprintf(stderr, "ioctl(PIN_CUDA dst) failed: ret=%d errno=%d\n", ret, errno);
		return 1;
	}

#if (SURFACE_W < 16) || (SURFACE_H < 16)
#error Grid and block sizes must be shrunk for small surfaces
#endif
#if (SURFACE_W & 15) || (SURFACE_H & 15)
#error Grid and block sizes are not a multiple of the surface size
#endif
	dim3 dimGrid(SURFACE_W / 16, SURFACE_H / 16);
	dim3 dimBlock(16, 16);
	fill_surface<<<dimGrid, dimBlock>>>(src_d, 0);
	fill_surface<<<dimGrid, dimBlock>>>(dst_d, 0xffffffffU);

	ce = hipDeviceSynchronize();
	if (ce != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize() failed: %d\n", ce);
		return 1;
	}

	dma_params.src = pin_params_src.handle;
	dma_params.dst = pin_params_dst.handle;
	dma_params.len = SURFACE_SIZE * sizeof(*src_d);
	dma_params.flags = PICOEVB_H2C2H_DMA_FLAG_SRC_IS_CUDA |
		PICOEVB_H2C2H_DMA_FLAG_DST_IS_CUDA;
	ret = ioctl(fd, PICOEVB_IOC_H2C2H_DMA, &dma_params);
	if (ret != 0) {
		fprintf(stderr, "ioctl(DMA) failed: %d\n", ret);
		return 1;
	}

	/*
	 * dGPU on x86 does not allow GPUDirect RDMA on host pinned memory
	 * (hipMalloc), so we must allocate device memory, and manually copy
	 * it to the host for validation.
	 */
#ifdef NV_BUILD_DGPU
	ce = hipHostMalloc(&dst_cpu, SURFACE_SIZE * sizeof(*dst_cpu), 0);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipHostMalloc(dst_cpu) failed\n");
		return 1;
	}
	ce = hipMemcpy(dst_cpu, dst_d, SURFACE_SIZE * sizeof(*dst_cpu), hipMemcpyDeviceToHost);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipMemcpy() failed: %d\n", ce);
		return 1;
	}
#else
	dst_cpu = dst_d;
#endif

	ret = 0;
	for (y = 0; y < SURFACE_H; y++) {
		for (x = 0; x < SURFACE_W; x++) {
			uint32_t expected = DATA(x, y);
			uint32_t offset = OFFSET(x, y);
			uint32_t actual = dst_cpu[offset];
			if (actual != expected) {
				fprintf(stderr,
					"dst[0x%x] is 0x%x not 0x%x\n",
					offset, actual, expected);
				ret = 1;
			}
		}
	}
	if (ret)
		return 1;

#ifdef NV_BUILD_DGPU
	ce = hipHostFree(dst_cpu);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipHostFree(dst_cpu) failed: %d\n", ce);
		return 1;
	}
#endif

	unpin_params_dst.handle = pin_params_dst.handle;
	ret = ioctl(fd, PICOEVB_IOC_UNPIN_CUDA, &unpin_params_dst);
	if (ret != 0) {
		fprintf(stderr, "ioctl(UNPIN_CUDA dst) failed: %d\n", ret);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipFree(dst_d);
#else
	ce = hipHostFree(dst_d);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Free of dst_d failed: %d\n", ce);
		return 1;
	}

	unpin_params_src.handle = pin_params_src.handle;
	ret = ioctl(fd, PICOEVB_IOC_UNPIN_CUDA, &unpin_params_src);
	if (ret != 0) {
		fprintf(stderr, "ioctl(UNPIN_CUDA src) failed: %d\n", ret);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipFree(src_d);
#else
	ce = hipHostFree(src_d);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Free of src_d failed: %d\n", ce);
		return 1;
	}

	ret = close(fd);
	if (ret < 0) {
		perror("close() failed");
		return 1;
	}

	return 0;
}
